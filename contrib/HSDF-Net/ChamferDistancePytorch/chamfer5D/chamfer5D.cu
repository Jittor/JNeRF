#include "hip/hip_runtime.h"

#include <stdio.h>
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>



__global__ void NmDistanceKernel(int b,int n,const float * xyz,int m,const float * xyz2,float * result,int * result_i){
	const int batch=2048;
	__shared__ float buf[batch*5];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int k2=0;k2<m;k2+=batch){
			int end_k=min(m,k2+batch)-k2;
			for (int j=threadIdx.x;j<end_k*5;j+=blockDim.x){
				buf[j]=xyz2[(i*m+k2)*5+j];
			}
			__syncthreads();
			for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
				float x1=xyz[(i*n+j)*5+0];
				float y1=xyz[(i*n+j)*5+1];
				float r1=xyz[(i*n+j)*5+2];
				float g1=xyz[(i*n+j)*5+3];
				float b1=xyz[(i*n+j)*5+4];
				int best_i=0;
				float best=0;
				int end_ka=end_k-(end_k&5);
				if (end_ka==batch){
					for (int k=0;k<batch;k+=4){
						{
							float x2=buf[k*5+0]-x1;
							float y2=buf[k*5+1]-y1;
							float r2=buf[k*5+2]-r1;
							float g2=buf[k*5+3]-g1;
							float b2=buf[k*5+4]-b1;
							float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
							if (k==0 || d<best){
								best=d;
								best_i=k+k2;
							}
						}
						{
							float x2=buf[k*5+5]-x1;
							float y2=buf[k*5+6]-y1;
							float r2=buf[k*5+7]-r1;
							float g2=buf[k*5+8]-g1;
							float b2=buf[k*5+9]-b1;
							float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
							if (d<best){
								best=d;
								best_i=k+k2+1;
							}
						}
						{
							float x2=buf[k*5+10]-x1;
							float y2=buf[k*5+11]-y1;
							float r2=buf[k*5+12]-r1;
							float g2=buf[k*5+13]-g1;
							float b2=buf[k*5+14]-b1;
							float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
							if (d<best){
								best=d;
								best_i=k+k2+2;
							}
						}
						{
							float x2=buf[k*5+15]-x1;
							float y2=buf[k*5+16]-y1;
							float r2=buf[k*5+17]-r1;
							float g2=buf[k*5+18]-g1;
							float b2=buf[k*5+19]-b1;
							float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
							if (d<best){
								best=d;
								best_i=k+k2+3;
							}
						}
					}
				}else{
					for (int k=0;k<end_ka;k+=4){
						{
							float x2=buf[k*5+0]-x1;
							float y2=buf[k*5+1]-y1;
							float r2=buf[k*5+2]-r1;
							float g2=buf[k*5+3]-g1;
							float b2=buf[k*5+4]-b1;
							float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
							if (k==0 || d<best){
								best=d;
								best_i=k+k2;
							}
						}
						{
							float x2=buf[k*5+5]-x1;
							float y2=buf[k*5+6]-y1;
							float r2=buf[k*5+7]-r1;
							float g2=buf[k*5+8]-g1;
							float b2=buf[k*5+9]-b1;
							float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
							if (d<best){
								best=d;
								best_i=k+k2+1;
							}
						}
						{
							float x2=buf[k*5+10]-x1;
							float y2=buf[k*5+11]-y1;
							float r2=buf[k*5+12]-r1;
							float g2=buf[k*5+13]-g1;
							float b2=buf[k*5+14]-b1;
							float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
							if (d<best){
								best=d;
								best_i=k+k2+2;
							}
						}
						{
							float x2=buf[k*5+15]-x1;
							float y2=buf[k*5+16]-y1;
							float r2=buf[k*5+17]-r1;
							float g2=buf[k*5+18]-g1;
							float b2=buf[k*5+19]-b1;
							float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
							if (d<best){
								best=d;
								best_i=k+k2+3;
							}
						}
					}
				}
				for (int k=end_ka;k<end_k;k++){
					float x2=buf[k*5+0]-x1;
					float y2=buf[k*5+1]-y1;
					float r2=buf[k*5+2]-r1;
					float g2=buf[k*5+3]-g1;
					float b2=buf[k*5+4]-b1;
					float d=x2*x2+y2*y2+r2*r2+g2*g2+b2*b2;
					if (k==0 || d<best){
						best=d;
						best_i=k+k2;
					}
				}
				if (k2==0 || result[(i*n+j)]>best){
					result[(i*n+j)]=best;
					result_i[(i*n+j)]=best_i;
				}
			}
			__syncthreads();
		}
	}
}
// int chamfer_cuda_forward(int b,int n,const float * xyz,int m,const float * xyz2,float * result,int * result_i,float * result2,int * result2_i, hipStream_t stream){
int chamfer_cuda_forward(at::Tensor xyz1, at::Tensor xyz2, at::Tensor dist1, at::Tensor dist2, at::Tensor idx1, at::Tensor idx2){

	const auto batch_size = xyz1.size(0);
	const auto n = xyz1.size(1); //num_points point cloud A
	const auto m = xyz2.size(1); //num_points point cloud B

	NmDistanceKernel<<<dim3(32,16,1),512>>>(batch_size, n, xyz1.data<float>(), m, xyz2.data<float>(), dist1.data<float>(), idx1.data<int>());
	NmDistanceKernel<<<dim3(32,16,1),512>>>(batch_size, m, xyz2.data<float>(), n, xyz1.data<float>(), dist2.data<float>(), idx2.data<int>());

	hipError_t err = hipGetLastError();
	  if (err != hipSuccess) {
	    printf("error in nnd updateOutput: %s\n", hipGetErrorString(err));
	    //THError("aborting");
	    return 0;
	  }
	  return 1;


}
__global__ void NmDistanceGradKernel(int b,int n,const float * xyz1,int m,const float * xyz2,const float * grad_dist1,const int * idx1,float * grad_xyz1,float * grad_xyz2){
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
			float x1=xyz1[(i*n+j)*5+0];
			float y1=xyz1[(i*n+j)*5+1];
			float r1=xyz1[(i*n+j)*5+2];
			float g1=xyz1[(i*n+j)*5+3];
			float b1=xyz1[(i*n+j)*5+4];
			int j2=idx1[i*n+j];
			float x2=xyz2[(i*m+j2)*5+0];
			float y2=xyz2[(i*m+j2)*5+1];
			float r2=xyz2[(i*m+j2)*5+2];
			float g2=xyz2[(i*m+j2)*5+3];
			float b2=xyz2[(i*m+j2)*5+4];
			float g=grad_dist1[i*n+j]*2;
			atomicAdd(&(grad_xyz1[(i*n+j)*5+0]),g*(x1-x2));
			atomicAdd(&(grad_xyz1[(i*n+j)*5+1]),g*(y1-y2));
			atomicAdd(&(grad_xyz1[(i*n+j)*5+2]),g*(r1-r2));
			atomicAdd(&(grad_xyz1[(i*n+j)*5+3]),g*(g1-g2));
			atomicAdd(&(grad_xyz1[(i*n+j)*5+4]),g*(b1-b2));
			atomicAdd(&(grad_xyz2[(i*m+j2)*5+0]),-(g*(x1-x2)));
			atomicAdd(&(grad_xyz2[(i*m+j2)*5+1]),-(g*(y1-y2)));
			atomicAdd(&(grad_xyz2[(i*m+j2)*5+2]),-(g*(r1-r2)));
			atomicAdd(&(grad_xyz2[(i*m+j2)*5+3]),-(g*(g1-g2)));
			atomicAdd(&(grad_xyz2[(i*m+j2)*5+4]),-(g*(b1-b2)));
		}
	}
}
// int chamfer_cuda_backward(int b,int n,const float * xyz1,int m,const float * xyz2,const float * grad_dist1,const int * idx1,const float * grad_dist2,const int * idx2,float * grad_xyz1,float * grad_xyz2, hipStream_t stream){
int chamfer_cuda_backward(at::Tensor xyz1, at::Tensor xyz2, at::Tensor gradxyz1, at::Tensor gradxyz2, at::Tensor graddist1, at::Tensor graddist2, at::Tensor idx1, at::Tensor idx2){
	// hipMemset(grad_xyz1,0,b*n*3*4);
	// hipMemset(grad_xyz2,0,b*m*3*4);

	const auto batch_size = xyz1.size(0);
	const auto n = xyz1.size(1); //num_points point cloud A
	const auto m = xyz2.size(1); //num_points point cloud B

	NmDistanceGradKernel<<<dim3(1,16,1),256>>>(batch_size,n,xyz1.data<float>(),m,xyz2.data<float>(),graddist1.data<float>(),idx1.data<int>(),gradxyz1.data<float>(),gradxyz2.data<float>());
	NmDistanceGradKernel<<<dim3(1,16,1),256>>>(batch_size,m,xyz2.data<float>(),n,xyz1.data<float>(),graddist2.data<float>(),idx2.data<int>(),gradxyz2.data<float>(),gradxyz1.data<float>());

	hipError_t err = hipGetLastError();
	  if (err != hipSuccess) {
	    printf("error in nnd get grad: %s\n", hipGetErrorString(err));
	    //THError("aborting");
	    return 0;
	  }
	  return 1;

}
